
#include <hip/hip_runtime.h>
__device__ __forceinline__ float sigmoid (float a)
{
    return 1.0 / (1.0 + exp (-a));
}

extern "C"
__global__ void sigmoid_kernel (const float * __restrict__ src, 
                                float * __restrict__ dst, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<n)
    {
        dst[i] = sigmoid(src[i]);
    }
}  